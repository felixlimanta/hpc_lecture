
#include <hip/hip_runtime.h>
#include <cstdio>

const int M = 32;

__global__ void bucket_sort(int *key, int *bucket, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) return;

  atomicAdd(bucket + key[i], 1); // Fill bucket by key
  __syncthreads(); // Wait for all threads to finish filling bucket

  for (int j = 0, k = 0; j <= i; k++) { // Output sorted data to key
    key[i] = k;
    j += bucket[k];
  }
}

int main() {
  int n = 50;
  int range = 5;

  int* key;
  hipMallocManaged(&key, n * sizeof(int));
  for (int i = 0; i < n; ++i) {
    key[i] = rand() % range;
    printf("%d ", key[i]);
  }
  printf("\n");

  int* bucket;
  hipMallocManaged(&bucket, range * sizeof(int));

  bucket_sort<<<(n + M - 1) / M, M, range>>>(key, bucket, n);
  hipDeviceSynchronize();

  for (int i = 0; i < n; i++) {
    printf("%d ", key[i]);
  }
  printf("\n");
}
